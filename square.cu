
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void square(const float *A, float *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        B[i] = A[i] * A[i];
    }
}

int main(void)
{

    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    std::cout << "[Vector addition of " <<  numElements << " elements]\n";

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
    }



    float *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid 
                << " blocks of " << threadsPerBlock << " threads\n";
    
    square<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements);

    hipDeviceSynchronize();//wait for all threads to be finished

    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);




    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] * h_A[i] - h_B[i]) > 1e-5)
        {
            std::cerr << "Result verification failed at element " << i << "!\n";
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED\n";
    hipFree(d_A);
    hipFree(d_B);

    free(h_A);
    free(h_B);

    std::cout << "Done\n";
    return 0;
}

